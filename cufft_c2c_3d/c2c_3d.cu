#include "hip/hip_runtime.h"

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<complex>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) 
{
	printf("[simpleCUFFT] is starting...\n");

	findCudaDevice(argc, (const char **)argv);
  
	const size_t Nx = (argc < 2) ? 100 : atoi(argv[1]);
	const size_t Ny = (argc < 3) ? 100 : atoi(argv[2]);
	const size_t Nz = (argc < 4) ? 100 : atoi(argv[3]);
    const unsigned int IsProf = (argc < 5) ? 0 : atoi(argv[4]);
	printf("Nx = %zu, Ny = %zu, Nz = %zu, IsProf = %d\n", Nx, Ny, Nz, IsProf);

    std::vector<float2> cx(Nx * Ny * Nz);
    std::vector<float2> cy(Nx * Ny * Nz);
    std::vector<float2> backx(cx.size());
	for(size_t i = 0; i < Nx; ++i)
	{
		for(size_t j = 0; j < Ny; ++j)
		{
			for(size_t k = 0; k < Nz; ++k)
			{
				const size_t pos = i * Ny * Nz + j * Nz + k;
				cx[pos].x = (i + j + k) * 1.0f;
				cx[pos].y = (i + j + k) * -0.1f;
			}
		}
	}
  
    // Create HIP device objects:
    hipfftComplex * x = NULL;
    hipfftComplex * y = NULL;
    hipMalloc(&x, cx.size() * sizeof(decltype(cx)::value_type));
	hipMalloc(&y, cy.size() * sizeof(decltype(cy)::value_type));
    hipMemcpy(x, cx.data(), cx.size() * sizeof(decltype(cx)::value_type), hipMemcpyHostToDevice);

    // Create plans
	hipfftHandle plan;
	hipfftHandle plan2;
	checkCudaErrors(hipfftPlan3d(&plan,  Nx, Ny, Nz, HIPFFT_C2C));	checkCudaErrors(hipfftExecC2C(plan,  x, y, HIPFFT_FORWARD));
	checkCudaErrors(hipfftPlan3d(&plan2, Nx, Ny, Nz, HIPFFT_C2C));	checkCudaErrors(hipfftExecC2C(plan2, y, x, HIPFFT_BACKWARD));  
	//checkCudaErrors(hipfftPlan3d(&plan, Nx * Ny * Nz, HIPFFT_D2Z));	checkCudaErrors(hipfftExecD2Z(plan, x, reinterpret_cast<hipfftDoubleComplex *>(y)));
	//checkCudaErrors(hipfftPlan3d(&plan2, Nx * Ny * Nz, HIPFFT_Z2D));checkCudaErrors(hipfftExecZ2D(plan2, reinterpret_cast<hipfftDoubleComplex *>(y), x));  
    hipMemcpy(backx.data(), x, backx.size() * sizeof(decltype(backx)::value_type), hipMemcpyDeviceToHost);
  

    double error = 0.0f;
    for(size_t i = 0; i < Nx; i++)
    {
        for(size_t j = 0; j < Ny; j++)
        {
            for(size_t k = 0; k < Nz; k++)
            {
				const size_t pos = i * Ny * Nz + j * Nz + k;
				double diffx = std::abs(backx[pos].x / (Nx*Ny*Nz) - cx[pos].x);
				double diffy = std::abs(backx[pos].y / (Nx*Ny*Nz) - cx[pos].y);
				double diff = diffx + diffy;
				if(diff > error)
					error = diff;
            }
        }
    }
    std::cout << "Maximum error: " << error << "\n";
	
	if(IsProf > 0)
	{
		int iteration_times = 1000;
		timespec startTime,stopTime;	
		double ElapsedMilliSec = 0;
		double ElapsedNanoSec = 0;
		clock_gettime(CLOCK_MONOTONIC, &startTime);
		for(int i = 0;i<iteration_times;i++)
			hipfftExecC2C(plan, x, y, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_MONOTONIC, &stopTime);
		double d_startTime = static_cast<double>(startTime.tv_sec)*1e9 + static_cast<double>(startTime.tv_nsec);
		double d_currentTime = static_cast<double>(stopTime.tv_sec)*1e9 + static_cast<double>(stopTime.tv_nsec);
		ElapsedNanoSec = d_currentTime - d_startTime;
		ElapsedMilliSec = ElapsedNanoSec / 1e6;
		printf("elapsed mill sec = %.3f(ms)\n", ElapsedMilliSec/iteration_times);
	}
	
	//free(cx);
	//free(backx);
	checkCudaErrors(hipFree(x));
	checkCudaErrors(hipFree(y));
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(plan2));
	
}

int main(int argc, char **argv) { runTest(argc, argv); }
