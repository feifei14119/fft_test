#include "hip/hip_runtime.h"

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<complex>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


////////////////////////////////////////////////////////////////////////////////
#define T float
#define T2 float2
void runTest(int argc, char **argv) 
{
	printf("[simpleCUFFT] is starting...\n");

	findCudaDevice(argc, (const char **)argv);
  
	const size_t Nx      = 100;
	const size_t Ny      = 100;
	const size_t Nz      = 100;

    std::vector<T> cx(Nx * Ny * Nz);
    std::vector<T> backx(cx.size());
    std::fill(cx.begin(), cx.end(), 0.0);
    for(size_t i = 0; i < Nx; ++i)
    {
        for(size_t j = 0; j < Ny; ++j)
        {
            for(size_t k = 0; k < Nz; ++k)
            {
                const size_t pos = i * Ny * Nz + j * Nz + k;
                cx[pos]          = i + j + k;
            }
        }
    }

    // Output buffer
    std::vector<std::complex<T>> cy(Nx * Ny * Nz);
  
    // Create HIP device objects:
    T* x = NULL;
    T2 * y = NULL;
    hipMalloc(&x, cx.size() * sizeof(decltype(cx)::value_type));
	hipMalloc(&y, cy.size() * sizeof(decltype(cy)::value_type));
    hipMemcpy(x, cx.data(), cx.size() * sizeof(decltype(cx)::value_type), hipMemcpyHostToDevice);


    // Create plans
	hipfftHandle plan;
	hipfftHandle plan2;
	checkCudaErrors(hipfftPlan3d(&plan, Nx, Ny, Nz, HIPFFT_R2C));			checkCudaErrors(hipfftExecR2C(plan, x, reinterpret_cast<hipfftComplex *>(y)));
	checkCudaErrors(hipfftPlan3d(&plan2, Nx, Ny, Nz, HIPFFT_C2R));		checkCudaErrors(hipfftExecC2R(plan2, reinterpret_cast<hipfftComplex *>(y), x));  
	//checkCudaErrors(hipfftPlan1d(&plan, Nx * Ny * Nz, HIPFFT_D2Z, 1));	checkCudaErrors(hipfftExecD2Z(plan, x, reinterpret_cast<hipfftDoubleComplex *>(y)));
	//checkCudaErrors(hipfftPlan1d(&plan2, Nx * Ny * Nz, HIPFFT_Z2D, 1));	checkCudaErrors(hipfftExecZ2D(plan2, reinterpret_cast<hipfftDoubleComplex *>(y), x));  
    hipMemcpy(backx.data(), x, backx.size() * sizeof(decltype(backx)::value_type), hipMemcpyDeviceToHost);
  

    double       error = 0.0f;
    for(size_t i = 0; i < Nx; i++)
    {
        for(size_t j = 0; j < Ny; j++)
        {
            for(size_t k = 0; k < Nz; k++)
            {
                double diff = std::abs(backx[i] / (Nx * Ny * Nz) - cx[i]);
                if(diff > error)
                    error = diff;
            }
        }
    }
    std::cout << "Maximum error: " << error << "\n";
	
	
	//free(cx);
	//free(backx);
	//checkCudaErrors(hipFree(x));
	//checkCudaErrors(hipFree(y));
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipfftDestroy(plan2));
	
}

int main(int argc, char **argv) { runTest(argc, argv); }
